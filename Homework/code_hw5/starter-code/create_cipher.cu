#include "hip/hip_runtime.h"
#include <algorithm>
#include <cctype>
#include <fstream>
#include <iostream>
#include <vector>
#include <numeric>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
// You may include other thrust headers if necessary.

#include "test_macros.h"

// You will need to call these functors from
// thrust functions in the code do not create new ones

// returns true if the char is not a lowercase letter
struct isnot_lowercase_alpha : thrust::unary_function<unsigned char, bool>
{
    __host__ __device__
    bool operator()(const unsigned char& x) const
    {
      return islower(x);
    }
};

// convert an uppercase letter into a lowercase one
// do not use the builtin C function or anything from boost, etc.
struct upper_to_lower : thrust::unary_function<unsigned char, unsigned char>
{
    __host__ __device__
    unsigned char operator()(const unsigned char& x) const
    {
      return x+32;
    }
};

// apply a shift with appropriate wrapping
struct apply_shift : thrust::binary_function<unsigned char, int, unsigned char>
{
    const unsigned char* shifts;
    const int period;

    apply_shift(unsigned char* shifts_, int& period_): period(period_){
      shifts = shifts_;
    }

    __host__ __device__
    unsigned char operator()(const unsigned char& x, const int& loc){
      return x + *(shifts+(loc%period));
    }
};

// Returns a vector with the top 5 letter frequencies in text.
std::vector<double> getLetterFrequencyCpu(const std::vector<unsigned char> &text)
{
    std::vector<unsigned int> freq(256);

    for (unsigned char c : text)
        freq[tolower(c)]++;

    unsigned int sum_chars = std::accumulate(
        freq.begin() + 'a',
        freq.begin() + 'z' + 1,
        0,
        std::plus<unsigned int>()
    );

    std::vector<double> freq_alpha_lower;

    for (unsigned char c = 'a'; c <= 'z'; ++c)
        if (freq[c] > 0)
            freq_alpha_lower.push_back(freq[c] / static_cast<double>(sum_chars));

    // pick the 5 most commonly occurring letters
    std::sort(freq_alpha_lower.begin(), freq_alpha_lower.end(), std::greater<double>());
    freq_alpha_lower.resize(std::min(static_cast<int>(freq_alpha_lower.size()), 5));
    return freq_alpha_lower;
}

// Print the top 5 letter frequencies and them.
std::vector<double> getLetterFrequencyGpu(const thrust::device_vector<unsigned char> &text)
{
    std::vector<double> freq_alpha_lower;
    // WARNING: make sure you handle the case of not all letters appearing
    // in the text.

    thrust::device_vector<unsigned char> text_sorted(text.size());
    thrust::copy(text.begin(), text.end(), text_sorted.begin());

    thrust::sort(text_sorted.begin(), text_sorted.end());
    thrust::device_vector<unsigned char> in_vals(256);
    thrust::device_vector<unsigned char> out_keys(256);
    thrust::device_vector<unsigned int> out_counts(256);

    thrust::fill(in_vals.begin(), in_vals.end(), 1);
    thrust::fill(out_counts.begin(), out_counts.end(), 0);

    auto new_end = thrust::reduce_by_key(text.begin(), text.end(),
      in_vals.begin(), out_keys.begin(), out_counts.begin());

    auto first = thrust::make_zip_iterator(
      thrust::make_tuple(out_counts.begin(), out_keys.begin())
    );

    auto last = thrust::make_zip_iterator(
      thrust::make_tuple(new_end.second, new_end.first)
    );

    return freq_alpha_lower;
}

int main(int argc, char **argv)
{
    if (argc != 3)
    {
        std::cerr << "Didn't supply plain text and period!" << std::endl;
        return 1;
    }

    // open file and verify
    std::ifstream ifs(argv[1], std::ios::binary);
    if (!ifs.good())
    {
        std::cerr << "Couldn't open book file!" << std::endl;
        return 1;
    }

    unsigned int period = std::atoi(argv[2]);
    if (period < 4)
    {
        std::cerr << "Period must be at least 4!" << std::endl;
        return 1;
    }

    std::vector<unsigned char> text;

    ifs.seekg(0, std::ios::end); // seek to end of file
    int length = ifs.tellg();    // get distance from beginning
    ifs.seekg(0, std::ios::beg); // move back to beginning

    text.resize(length);
    ifs.read((char *) &text[0], length);

    ifs.close();

    thrust::device_vector<unsigned char> text_clean;
    // TODO: sanitize input to contain only a-z lowercase (use the
    // isnot_lowercase_alpha functor), calculate the number of characters
    // in the cleaned text and put the result in text_clean, make sure to
    // resize text_clean to the correct size!
    int numElements = -1;

    std::cout << std::endl << "Before ciphering!" << std::endl << std::endl;
    std::vector<double> letterFreqGpu = getLetterFrequencyGpu(text_clean);
    std::vector<double> letterFreqCpu = getLetterFrequencyCpu(text);
    bool success = true;
    EXPECT_VECTOR_EQ_EPS(letterFreqCpu, letterFreqGpu, 1e-14, &success);
    PRINT_SUCCESS(success);

    thrust::device_vector<unsigned int> shifts(period);
    // TODO fill in shifts using thrust random number generation (make sure
    // not to allow 0-shifts, this would make for rather poor encryption).

    std::cout << std::endl << "Encryption key: ";

    for (unsigned int i = 0; i < period; ++i)
        std::cout << static_cast<char>('a' + shifts[i]);
    std::cout << std::endl;

    thrust::device_vector<unsigned char> device_cipher_text(numElements);

    // TODO: Apply the shifts to text_clean and place the result in
    // device_cipher_text.

    thrust::host_vector<unsigned char> host_cipher_text = device_cipher_text;

    std::cout << std::endl << "After ciphering!" << std::endl << std::endl;
    getLetterFrequencyGpu(device_cipher_text);

    std::ofstream ofs("cipher_text.txt", std::ios::binary);
    ofs.write((char *) &host_cipher_text[0], numElements);

    ofs.close();
    return 0;
}
