#include "hip/hip_runtime.h"
#include "gpu_func.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <iostream>
#include "hipblas.h"

#define BLOCK_SIZE 32

__global__
void device_add_one(int* d_result, int t) {
    *d_result = t + 1;
}

/*
Just a dummy function that can be used to warm up GPU
*/
int useless_gpu_add_one(int t) {
    int result;
    int* d_result;

    checkCudaErrors(hipMalloc((void**)&d_result, 1 * sizeof(int)));

    event_pair timer;
    start_timer(&timer);
    device_add_one<<<1,1>>>(d_result, t);
    check_launch("device_add_one");
    double time = stop_timer(&timer);

    std::cout << "device_add_one took: " << time << " seconds" << std::endl;

    checkCudaErrors(hipMemcpy(&result, d_result, 1 * sizeof(int),
                               hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(d_result));
    return result;
}

// __global__
// void gemm_gpu(double* A, double* B, double* C, double* D, int hA, int wA,
//     int hB, int wB, double alpha, double beta){
//   int bx = blockIdx.x;
//   int by = blockIdx.y;
//
//   int tx = threadIdx.x;
//   int ty = threadIdx.y;
//
//   int a_start = wA*BLOCK_SIZE*by;
//   int a_end   = a_start+wA-1;
//   int a_step  = BLOCK_SIZE;
//
//   int b_start = BLOCK_SIZE*bx*wB;
//   int b_step  = BLOCK_SIZE*wB;
//
//   float Dsub = 0;
//
//   for (int a = a_start, b=b_start; a <= a_end; a+=a_step, b+=b_step){
//
//     __shared__ float Asub[BLOCK_SIZE][BLOCK_SIZE];
//     __shared__ float Bsub[BLOCK_SIZE][BLOCK_SIZE];
//
//     if (bx*BLOCK_SIZE+tx >= wA || bx*BLOCK_SIZE+ty >= hB ||
//       by*BLOCK_SIZE+ty >= hA || by*BLOCK_SIZE+tx >= wB){
//       Asub[ty][tx] = 0;
//       Bsub[ty][tx] = 0;
//     }
//     else{
//       //printf("a: %u %u %u %u\n", a,tx,ty,a + tx + wA*ty);
//       //printf("b: %u %u %u %u\n", b,tx,ty,b + tx + wA*ty);
//       Asub[ty][tx] = A[a + tx + wA*ty];
//       Bsub[ty][tx] = B[b + tx + wB*ty];
//     }
//
//     __syncthreads();
//
//     for (int i = 0; i < BLOCK_SIZE; i++){
//       Dsub += Asub[ty][i]*Bsub[i][tx];
//     }
//     __syncthreads();
//   }
//
//   if (bx*BLOCK_SIZE+tx < wB && by*BLOCK_SIZE+ty < hA){
//     int c = wB*BLOCK_SIZE*by+BLOCK_SIZE*bx;
//     //printf("c: %u %u %u %u\n", c,tx,ty,c + tx + ty*wB);
//     D[c + tx + ty*wB] = alpha*Dsub + beta*C[c + tx + ty*wB];
//   }
//
// }

__global__
void gemm_gpu(double* A, double* B, double* C, double* D, int hA, int wA,
    int hB, int wB, double alpha, double beta){
  int bx = blockIdx.x;
  int by = blockIdx.y;

  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int row = by*BLOCK_SIZE+ty;
  int col = bx*BLOCK_SIZE+tx;

  int a_start = row;
  int a_step  = hA;
  int b_start = col*hB;

  double entry = 0;

  if (row < hA && col < wB){
    for (int i = 0; i < hB; i++){
      entry += A[a_start+i*a_step]*B[b_start+i];
    }
    D[row+col*hA] = alpha*entry + beta*C[row+col*hA];
  }

}

/*
Routine to perform an in-place GEMM operation, i.e., C := alpha*A*B + beta*C
*/
int myGEMM(double* __restrict__ A, double* __restrict__ B,
           double* __restrict__ C, double* alpha, double* beta,
           int M, int N, int K) {
    /* TODO: Write an efficient GEMM implementation on GPU */


    int c_size = M*N*sizeof(double);

    double* Dd;
    hipMalloc((void**)&Dd, c_size);

    dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
    dim3 dimGrid(N/dimBlock.x+1, M/dimBlock.y+1);

    gemm_gpu<<<dimGrid, dimBlock>>>(A, B, C, Dd, M, K, K, N,
      *alpha, *beta);

    hipMemcpy(C, Dd, c_size, hipMemcpyDeviceToDevice);

    hipFree(Dd);
    return 0;
}
